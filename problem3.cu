#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void add_i(int * arr) { 
	arr[blockIdx.x] += blockIdx.x;
}


__global__ void initialize(int * arr) {
	arr[blockIdx.x] = 0;
	add_i(arr);
}

int main(int argc, char ** argv) {
	int * arr;
	int * d_arr;
	int n = 1024;
	int size = n * sizeof(int);
	
	arr = (int *)malloc(size);
	hipMalloc((void **) &d_arr, size);
	hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
	initialize<<<n, 1>>>(d_arr);
	hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < n; i++) {
		printf("%d ", arr[i]);
		if (i % 16 == 0 && i != 0) {
			printf("\n");
		}
	}
	printf("\n");
}
