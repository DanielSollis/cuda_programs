#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void initialize(int * arr) {
	arr[blockIdx.x] = 0;
}

int main(int argc, char ** argv) {
	int * arr;
	int * d_arr;
	int n = 1024;
	int size = n * sizeof(int);
	
	arr = (int *)malloc(size);
	hipMalloc((void **) &d_arr, size);
	hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
	initialize<<<n, 1>>>(d_arr);
	hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < n; i++) {
		printf("%d ", arr[i]);
	}
	printf("\n");
}
