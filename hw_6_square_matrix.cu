#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

const int N = 10;

void init_matrix(int * matrix, int size) {
	for (int i = 0; i < size; i++) {
                for (int j = 0; j < size; j++) {        
			matrix[i][j] = i + j;
			printf("%d ", matrix[i][j]);
                }
		printf("\n");
        }
	printf("\n\n");
}


void print_matrix(int * matrix, int size) {
	for (int i = 0; i < size; i++) {
                printf("%d", matrix[i]);
	}
}

__global__ void square(int * matrix, int * result, int size) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned i = id / size;
	unsigned j = id % size;
	for (unsigned k = 0; k < size; ++k) {
		result[i * size + j] += 
			matrix[i * size + k] * 
				matrix[k * size + j];
	}
}

int main(int argc, char ** argv) {
	int * matrix, * result;
	init_matrix(matrix, N);
	square<<<N, N>>>(matrix, result, N);
	print_matrix(matrix, N);
	return 0;
}
