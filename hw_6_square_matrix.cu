#include <stdio.h>
#include <hip/hip_runtime.h>

const int N = 10;

__global__ void square(int * matrix, int * result, int size) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned ii = id / size;
        unsigned jj = id % size;
        for (unsigned kk = 0; kk < size; ++kk) {
                result[ii * size + jj] +=
                        matrix[kk * size + ii] *
                                matrix[kk * size + jj];
        }
}


void init_matrix(int * matrix, int size) {
	for (int i = 0; i <= size; i++) {
		matrix[i] = i;
	}

	for (int i = 1; i < size; i++) {
		for (int j = i * size, k = i; j < i * size + size; j++, k++) {
			matrix[j] = k;
		}
	}
	printf("matrix initialized\n");
}

	
void print_matrix(int * matrix, int size) {
	printf("printing matrix:\n");
	for (int i = 0; i < size * size; i++) {
		if (i % size == 0 && i != 0) {
			printf("\n");
		}
		printf("%d\t", matrix[i]);
	}
	printf("\n");
}


int main(int argc, char ** argv) {
	int size = N * N;
	int space = sizeof(int) * size;
	
	int * matrix = (int *)malloc(space);
	int * result, * d_matrix, * d_result;
	
	hipMalloc((void **) &d_matrix, space);
	hipMalloc((void **) &d_result, space);
        hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, size, hipMemcpyHostToDevice);

	init_matrix(matrix, N);
	printf("initialized out\n");
	print_matrix(matrix, N);	
        
	square<<<N, N>>>(matrix, d_result, N);

        hipMemcpy(result, d_result, N, hipMemcpyDeviceToHost);

	print_matrix(result, N);
	return 0;
}
